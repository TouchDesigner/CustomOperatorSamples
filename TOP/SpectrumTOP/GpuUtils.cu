#include "hip/hip_runtime.h"
/* Shared Use License: This file is owned by Derivative Inc. (Derivative)
* and can only be used, and/or modified for use, in conjunction with
* Derivative's TouchDesigner software, and only if you are a licensee who has
* accepted Derivative's TouchDesigner license or assignment agreement
* (which also govern the use of this file). You may share or redistribute
* a modified version of this file provided the following conditions are met:
*
* 1. The shared file or redistribution must retain the information set out
* above and this list of conditions.
* 2. Derivative's name (Derivative Inc.) or its trademarks may not be used
* to endorse or promote products derived from this file without specific
* prior written permission from Derivative.
*/
#include "GpuUtils.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"

#include <opencv2/core/cuda.hpp>

namespace
{
	void 
	createSurfaceObj(hipSurfaceObject_t* sObj, hipArray* arr)
	{
		hipResourceDesc resDesc = {};
		resDesc.res.array.array = arr;
		resDesc.resType = hipResourceTypeArray;
		hipCreateSurfaceObject(sObj, &resDesc);
	}

	__global__ void
	copySurfaceToMat(hipSurfaceObject_t src,
		uchar* dst, size_t dstStep,
		int width, int height)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			uchar data;
			// Read from input surface
			surf2Dread(&data, src, x, y);
			// Write to output mat data
			uchar* pixel = dst + (height - y - 1) * dstStep + x;
			*pixel = data;
		}
	}

	// Assume src is in 32F and dst is 32FC2
	__global__ void
	copy32FSurfaceToComplexMat(hipSurfaceObject_t src,
		uchar* dst, size_t dstStep,
		size_t width, size_t height, size_t xOffset, size_t pixelSize)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			float2 data{};
			// Read from input surface
			surf2Dread(&data.x, src, x * pixelSize + xOffset, y);
			// Write to output mat data
			float2* pixel = reinterpret_cast<float2*>(dst + (height - y - 1) * dstStep + x * 2 * sizeof(float));
			*pixel = data;
		}
	}

	// Assume src is in 16F and dst is 32FC2
	__global__ void
	copy16FSurfaceToComplexMat(hipSurfaceObject_t src,
		uchar* dst, size_t dstStep,
		size_t width, size_t height, size_t xOffset, size_t pixelSize)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			float2 data{};
			// Read from input surface
			half x16bits;
			surf2Dread(reinterpret_cast<uint16_t*>(&x16bits), src, x * pixelSize + xOffset, y);
			data.x = x16bits;
			// Write to output mat data
			float2* pixel = reinterpret_cast<float2*>(dst + (height - y - 1) * dstStep + x * 2 * sizeof(float));
			*pixel = data;
		}
	}

	// Assume src is in 16U and dst is 32FC2
	__global__ void
	copy16USurfaceToComplexMat(hipSurfaceObject_t src,
		uchar* dst, size_t dstStep,
		size_t width, size_t height, size_t xOffset, size_t pixelSize)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			float2 data{};
			// Read from input surface
			uint16_t x16bits;
			surf2Dread(&x16bits, src, x * pixelSize + xOffset, y);
			data.x = x16bits * 1.0 / 65536.0f;
			// Write to output mat data
			float2* pixel = reinterpret_cast<float2*>(dst + (height - y - 1) * dstStep + x * 2 * sizeof(float));
			*pixel = data;
		}
	}

	// Assume src is in 8U and dst is 32FC2
	__global__ void
	copy8USurfaceToComplexMat(hipSurfaceObject_t src,
			uchar* dst, size_t dstStep,
			size_t width, size_t height, size_t xOffset, size_t pixelSize)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			float2 data{};
			// Read from input surface
			uint8_t x8bit;
			surf2Dread(&x8bit, src, x * pixelSize + xOffset, y);
			data.x = x8bit * 1.0 / 256.0f;
			// Write to output mat data
			float2* pixel = reinterpret_cast<float2*>(dst + (height - y - 1) * dstStep + x * 2 * sizeof(float));
			*pixel = data;
		}
	}

	// Assumes input is 32FC2 and output 32FC1
	__global__ void
	copyComplexMatToSurface(uchar* src,
		hipSurfaceObject_t dst, size_t srcStep,
		int width, int height)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			// Read from mat data
			float* data = reinterpret_cast<float*>(src + (height - y - 1) * srcStep + x * 2 * sizeof(float));
			// Write to surface
			surf2Dwrite(*data, dst, x * sizeof(float), y);
		}
	}

	__global__ void
	copyMatToSurface(uchar* src,
		hipSurfaceObject_t dst, size_t srcStep,
		int width, int height)
	{
		// Calculate surface coordinates
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			// Read from mat data
			uchar data = *(src + (height - y - 1) * srcStep + x);
			// Write to surface
			surf2Dwrite(data, dst, x, y);
		}
	}
}

// input is float channels and output is CV_32FC2
void 
GpuUtils::arrayToComplexMatGPU(int width, int height, hipArray* input, cv::cuda::GpuMat& output, int numChannels, int channel, ChannelFormat cf)
{
	// Create the output surface object
	hipSurfaceObject_t inputS{};
	createSurfaceObj(&inputS, input);

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);

	switch (cf)
	{
	case GpuUtils::ChannelFormat::U8:
		copy8USurfaceToComplexMat<<<gridSize, blockSize>>>(inputS, output.data, output.step, width, height, channel * 1, numChannels * 1);
		break;
	case GpuUtils::ChannelFormat::U16:
		copy16USurfaceToComplexMat<<<gridSize, blockSize >> >(inputS, output.data, output.step, width, height, channel * 2, numChannels * 2);
		break;
	case GpuUtils::ChannelFormat::F16:
		copy16FSurfaceToComplexMat<<<gridSize, blockSize >> >(inputS, output.data, output.step, width, height, channel * 2, numChannels * 2);
		break;
	case GpuUtils::ChannelFormat::F32:
		copy32FSurfaceToComplexMat<<<gridSize, blockSize >> >(inputS, output.data, output.step, width, height, channel * 4, numChannels * 4);
		break;
	default:
		break;
	}

	hipDestroySurfaceObject(inputS);
}

void 
GpuUtils::complexMatGPUToArray(int width, int height, const cv::cuda::GpuMat& input, hipArray* output)
{
	// Create the output surface object
	hipSurfaceObject_t outputS{};
	createSurfaceObj(&outputS, output);

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);

	copyComplexMatToSurface<<<gridSize, blockSize>>>(input.data, outputS, input.step, width, height);

	hipDestroySurfaceObject(outputS);
}

void 
GpuUtils::matGPUToArray(int width, int height, const cv::cuda::GpuMat& input, hipArray* output, int pixelSize)
{
	width *= pixelSize;

	// Create the output surface object
	hipSurfaceObject_t outputS{};
	createSurfaceObj(&outputS, output);

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);

	uchar* inData = input.data;

	copyMatToSurface << <gridSize, blockSize >> >(inData, outputS, input.step, width, height);

	hipDestroySurfaceObject(outputS);
}

void 
GpuUtils::arrayToMatGPU(int width, int height, hipArray* input, cv::cuda::GpuMat& output, int pixelSize)
{
	width *= pixelSize;

	// Create the output surface object
	hipSurfaceObject_t inputS{};
	createSurfaceObj(&inputS, input);

	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);

	uchar* outData = output.data;

	copySurfaceToMat << <gridSize, blockSize >> >(inputS, outData, output.step, width, height);

	hipDestroySurfaceObject(inputS);
}